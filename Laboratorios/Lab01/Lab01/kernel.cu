#include "hip/hip_runtime.h"
﻿
#include ""

#include <stdio.h>
#include <stdlib.h>

__global__ void idx_calc_2d(int* input) {
    int tid = threadIdx.x;
    
    int row_offset = gridDim.x * blockDim.x * blockIdx.y;
    int block_offset = blockDim.x * blockIdx.x;
    int gid = tid + row_offset + block_offset;

    printf("[DEVICE] Global Id: %d\n", gid);
}

int main()
{
    // Inicialización
    dim3 blockSize(4, 4, 4);
    dim3 gridSize(2, 2, 2);

    int* c_cpu;
    int* a_cpu;
    int* b_cpu;

    int* c_device;
    int* a_device;
    int* b_device;

    const int data_count = 10000;
    const int data_size = data_count * sizeof(int);

    c_cpu = (int*)malloc(data_size);
    a_cpu = (int*)malloc(data_size);
    b_cpu = (int*)malloc(data_size);

    // Asignación de memoria en el dispositivo
    hipMalloc((void**)&c_device, data_size);
    hipMalloc((void**)&a_device, data_size);
    hipMalloc((void**)&b_device, data_size);

    // Transferir datos del host al dispositivo
    hipMemcpy(c_device, c_cpu, data_size, hipMemcpyHostToDevice);
    hipMemcpy(a_device, a_cpu, data_size, hipMemcpyHostToDevice);
    hipMemcpy(b_device, b_cpu, data_size, hipMemcpyHostToDevice);

    // Lanzar el kernel
    idx_calc_2d << <gridSize, blockSize >> > ();

    // Transferir datos del dispositivo al host
    hipMemcpy(c_cpu, c_device, data_size, hipMemcpyDeviceToHost);
    hipMemcpy(a_cpu, a_device, data_size, hipMemcpyDeviceToHost);
    hipMemcpy(b_cpu, b_device, data_size, hipMemcpyDeviceToHost);

    // Liberar memoria en el host
    free(c_cpu);
    free(a_cpu);
    free(b_cpu);

    // Liberar memoria en el dispositivo
    hipFree(c_device);
    hipFree(a_device);
    hipFree(b_device);

    // Restablecer el dispositivo CUDA
    hipDeviceReset();

    return 0;
}